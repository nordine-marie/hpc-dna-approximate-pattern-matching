#include "hip/hip_runtime.h"
/**
 * APPROXIMATE PATTERN MATCHING
 *
 * INF560 X2016
 */
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define APM_DEBUG 0

char *
read_input_file(char *filename, int *size)
{
    char *buf;
    off_t fsize;
    int fd = 0;
    int n_bytes = 1;

    /* Open the text file */
    fd = open(filename, O_RDONLY);
    if (fd == -1)
    {
        fprintf(stderr, "Unable to open the text file <%s>\n", filename);
        return NULL;
    }

    /* Get the number of characters in the textfile */
    fsize = lseek(fd, 0, SEEK_END);
    lseek(fd, 0, SEEK_SET);

    /* TODO check return of lseek */

#if APM_DEBUG
    printf("File length: %lld\n", fsize);
#endif

    /* Allocate data to copy the target text */
    buf = (char *)malloc(fsize * sizeof(char));
    if (buf == NULL)
    {
        fprintf(stderr, "Unable to allocate %ld byte(s) for main array\n", fsize);
        return NULL;
    }

    n_bytes = read(fd, buf, fsize);
    if (n_bytes != fsize)
    {
        fprintf(stderr, "Unable to copy %ld byte(s) from text file (%d byte(s) copied)\n", fsize, n_bytes);
        return NULL;
    }

#if APM_DEBUG
    printf("Number of read bytes: %d\n", n_bytes);
#endif

    *size = n_bytes;

    close(fd);

    return buf;
}

#define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

__global__ void cuda_levenshtein(char *s1, char *s2, int len, int *column, int *gpu_column)
{
    unsigned int x, y, lastdiag, olddiag;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (y = 1; y <= len; y++)
    {
        column[y] = y;
    }
    for (x = 1; x <= len; x++)
    {
        column[0] = x;
        lastdiag = x - 1;
        for (y = 1; y <= len; y++)
        {
            olddiag = column[y];
            column[y] = MIN3(
                column[y] + 1,
                column[y - 1] + 1,
                lastdiag + (s1[y - 1] == s2[x - 1+i] ? 0 : 1));
            lastdiag = olddiag;
        }
    }
    printf("coucou");
    gpu_column[i] = column[len];
}

int main(int argc, char **argv)
{
    char **pattern;
    char *filename;
    int approx_factor = 0;
    int nb_patterns = 0;
    int i;
    char *buf;
    struct timeval t1, t2;
    double duration;
    int n_bytes;
    int *n_matches;

    /* Check number of arguments */
    if (argc < 4)
    {
        printf("Usage: %s approximation_factor "
               "dna_database pattern1 pattern2 ...\n",
               argv[0]);
        return 1;
    }

    /* Get the distance factor */
    approx_factor = atoi(argv[1]);

    /* Grab the filename containing the target text */
    filename = argv[2];

    /* Get the number of patterns that the user wants to search for */
    nb_patterns = argc - 3;

    /* Fill the pattern array */
    pattern = (char **)malloc(nb_patterns * sizeof(char *));
    if (pattern == NULL)
    {
        fprintf(stderr,
                "Unable to allocate array of pattern of size %d\n",
                nb_patterns);
        return 1;
    }

    /* Grab the patterns */
    for (i = 0; i < nb_patterns; i++)
    {
        int l;

        l = strlen(argv[i + 3]);
        if (l <= 0)
        {
            fprintf(stderr, "Error while parsing argument %d\n", i + 3);
            return 1;
        }

        pattern[i] = (char *)malloc((l + 1) * sizeof(char));
        if (pattern[i] == NULL)
        {
            fprintf(stderr, "Unable to allocate string of size %d\n", l);
            return 1;
        }

        strncpy(pattern[i], argv[i + 3], (l + 1));
    }

    printf("Approximate Pattern Mathing: "
           "looking for %d pattern(s) in file %s w/ distance of %d\n",
           nb_patterns, filename, approx_factor);

    buf = read_input_file(filename, &n_bytes);
    if (buf == NULL)
    {
        return 1;
    }

    /* Allocate the array of matches */
    n_matches = (int *)malloc(nb_patterns * sizeof(int));
    if (n_matches == NULL)
    {
        fprintf(stderr, "Error: unable to allocate memory for %ldB\n",
                nb_patterns * sizeof(int));
        return 1;
    }

    /*****
   * BEGIN MAIN LOOP
   ******/

    /* Timer start */
    gettimeofday(&t1, NULL);
    int blocksize = 1024;
    int *gpu_column;
    char *gpu_pattern;
    char *gpu_buf;

    for (i = 0; i < nb_patterns; i++)
    {
        int size_pattern = strlen(pattern[i]);

        int *column;

        n_matches[i] = 0;

        column = (int *)malloc((size_pattern + 1) * sizeof(int));
        if (column == NULL)
        {
            fprintf(stderr, "Error: unable to allocate memory for column (%ldB)\n",
                    (size_pattern + 1) * sizeof(int));
            return 1;
        }
        hipMalloc(&gpu_column, (size_pattern + 1) * sizeof(int));
        hipMalloc(&gpu_pattern, (size_pattern) * sizeof(char));
        hipMalloc(&gpu_buf, n_bytes * sizeof(char));

        hipMemcpy(gpu_column, column, (size_pattern + 1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(gpu_pattern, pattern[i], (size_pattern+1) * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(gpu_buf, buf, (n_bytes * n_bytes) * sizeof(char), hipMemcpyHostToDevice);

        dim3 dimBlock(blocksize);
        dim3 dimGrid(ceil((size_pattern + 1) * sizeof(char) / (int)blocksize));

        cuda_levenshtein<<<dimGrid, dimBlock>>>(gpu_pattern, gpu_buf, size_pattern, column, gpu_column);
        hipDeviceSynchronize();
        
        printf(hipGetErrorString(hipPeekAtLastError()));

        hipMemcpy(column, gpu_column, (size_pattern + 1) * sizeof(int), hipMemcpyDeviceToHost);
        
        if (column[i] <= approx_factor)
        {
            n_matches[i]++;
        }
        free(column);
        hipFree(gpu_column);
        hipFree(gpu_buf);
        hipFree(gpu_pattern);
    }

    /* Timer stop */
    gettimeofday(&t2, NULL);

    duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);

    printf("APM done in %lf s\n", duration);

    /*****
   * END MAIN LOOP
   ******/

    for (i = 0; i < nb_patterns; i++)
    {
        printf("Number of matches for pattern <%s>: %d\n",
               pattern[i], n_matches[i]);
    }

    return 0;
}
