#include "hip/hip_runtime.h"
/**
 * APPROXIMATE PATTERN MATCHING
 *
 * INF560 X2016
 */
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/time.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define APM_DEBUG 0

char *
read_input_file(char *filename, int *size)
{
    char *buf;
    off_t fsize;
    int fd = 0;
    int n_bytes = 1;

    /* Open the text file */
    fd = open(filename, O_RDONLY);
    if (fd == -1)
    {
        fprintf(stderr, "Unable to open the text file <%s>\n", filename);
        return NULL;
    }

    /* Get the number of characters in the textfile */
    fsize = lseek(fd, 0, SEEK_END);
    lseek(fd, 0, SEEK_SET);

    /* TODO check return of lseek */

#if APM_DEBUG
    printf("File length: %lld\n", fsize);
#endif

    /* Allocate data to copy the target text */
    buf = (char *)malloc(fsize * sizeof(char));
    if (buf == NULL)
    {
        fprintf(stderr, "Unable to allocate %ld byte(s) for main array\n", fsize);
        return NULL;
    }

    n_bytes = read(fd, buf, fsize);
    if (n_bytes != fsize)
    {
        fprintf(stderr, "Unable to copy %ld byte(s) from text file (%d byte(s) copied)\n", fsize, n_bytes);
        return NULL;
    }

#if APM_DEBUG
    printf("Number of read bytes: %d\n", n_bytes);
#endif

    *size = n_bytes;

    close(fd);

    return buf;
}

#define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

__global__ void cuda_levenshtein(char *gpu_pattern, char *gpu_buf, int size_pattern, int n_bytes, int approx_factor, int *gpu_column, int *gpu_matches)
{
    unsigned int x, y, lastdiag, olddiag;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    gpu_column = &gpu_column[i * (size_pattern + 1)];
    gpu_buf = &gpu_buf[i];

    if (i < n_bytes)
    {
        int distance = 0;
        int size;
        size = size_pattern;
        if (n_bytes - i < size_pattern)
        {
            size = n_bytes - i;
        }

        for (y = 1; y <= size; y++)
        {
            gpu_column[y] = y;
        }
        for (x = 1; x <= size; x++)
        {
            gpu_column[0] = x;
            lastdiag = x - 1;
            for (y = 1; y <= size; y++)
            {
                olddiag = gpu_column[y];
                gpu_column[y] = MIN3(gpu_column[y] + 1, gpu_column[y - 1] + 1, lastdiag + (gpu_pattern[y - 1] == gpu_buf[x - 1] ? 0 : 1));
                lastdiag = olddiag;
            }
        }
        distance = gpu_column[size];

        if (distance <= approx_factor)
        {
            gpu_matches[i] = 1;
        }
        else
        {
            gpu_matches[i] = 0;
        }
    }
}

int main(int argc, char **argv)
{
    char **pattern;
    char *filename;
    int approx_factor = 0;
    int nb_patterns = 0;
    int i;
    char *buf;
    struct timeval t1, t2;
    double duration;
    int n_bytes;
    int *n_matches;

    /* Check number of arguments */
    if (argc < 4)
    {
        printf("Usage: %s approximation_factor "
               "dna_database pattern1 pattern2 ...\n",
               argv[0]);
        return 1;
    }

    /* Get the distance factor */
    approx_factor = atoi(argv[1]);

    /* Grab the filename containing the target text */
    filename = argv[2];

    /* Get the number of patterns that the user wants to search for */
    nb_patterns = argc - 3;

    /* Fill the pattern array */
    pattern = (char **)malloc(nb_patterns * sizeof(char *));
    if (pattern == NULL)
    {
        fprintf(stderr, "Unable to allocate array of pattern of size %d\n",
                nb_patterns);
        return 1;
    }

    /* Grab the patterns */
    for (i = 0; i < nb_patterns; i++)
    {
        int l;

        l = strlen(argv[i + 3]);
        if (l <= 0)
        {
            fprintf(stderr, "Error while parsing argument %d\n", i + 3);
            return 1;
        }

        pattern[i] = (char *)malloc((l + 1) * sizeof(char));
        if (pattern[i] == NULL)
        {
            fprintf(stderr, "Unable to allocate string of size %d\n", l);
            return 1;
        }

        strncpy(pattern[i], argv[i + 3], (l + 1));
    }

    printf("Approximate Pattern Mathing: "
           "looking for %d pattern(s) in file %s w/ distance of %d\n",
           nb_patterns, filename, approx_factor);

    buf = read_input_file(filename, &n_bytes);
    if (buf == NULL)
    {
        return 1;
    }

    /* Allocate the array of matches */
    n_matches = (int *)malloc(nb_patterns * sizeof(int));
    if (n_matches == NULL)
    {
        fprintf(stderr, "Error: unable to allocate memory for %ldB\n",
                nb_patterns * sizeof(int));
        return 1;
    }

    /*****
   * BEGIN MAIN LOOP
   ******/

    /* Timer start */
    gettimeofday(&t1, NULL);
        int blocksize = 1024;
        int nb_threads = min(blocksize,n_bytes) * ceil((n_bytes / (float)blocksize));
        dim3 dimBlock(min(blocksize,n_bytes));
        dim3 dimGrid(ceil((n_bytes / (float)blocksize)));

    for (i = 0; i < nb_patterns; i++)
    {

        int size_pattern = strlen(pattern[i]);

        int *nb_matches = (int *)malloc((n_bytes) * sizeof(int));

        for (int j = 0; j < n_bytes; j++)
        {
            n_matches[i] = 0;
        }

        n_matches[i] = 0;

        char *gpu_pattern;
        char *gpu_buf;
        int *gpu_matches;
        int *gpu_column;

        hipMalloc(&gpu_pattern, (size_pattern) * sizeof(char));
        hipMalloc(&gpu_buf, (n_bytes) * sizeof(char));
        hipMalloc(&gpu_matches, (n_bytes) * sizeof(int));
        hipMalloc(&gpu_column, nb_threads * (size_pattern + 1) * sizeof(int));

        hipMemcpy(gpu_pattern, pattern[i], (size_pattern) * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(gpu_buf, buf, (n_bytes) * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(gpu_matches, nb_matches, (n_bytes) * sizeof(int), hipMemcpyHostToDevice);

        cuda_levenshtein<<<dimGrid, dimBlock>>>(gpu_pattern, gpu_buf, size_pattern, n_bytes, approx_factor, gpu_column, gpu_matches);

        hipMemcpy(nb_matches, gpu_matches, (n_bytes) * sizeof(int), hipMemcpyDeviceToHost);

        for (int j = 0; j < n_bytes; j++)
        {
            n_matches[i] += nb_matches[j];
        }

        hipFree(gpu_pattern);
        hipFree(gpu_buf);
        hipFree(gpu_matches);
        hipFree(gpu_column);
    }

    /* Timer stop */
    gettimeofday(&t2, NULL);

    duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);

    printf("APM done in %lf s\n", duration);

    /*****
   * END MAIN LOOP
   ******/

    for (i = 0; i < nb_patterns; i++)
    {
        printf("Number of matches for pattern <%s>: %d\n", pattern[i],
               n_matches[i]);
    }

    return 0;
}